
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

// Задание 2: Сложение двух матриц
__global__
void ArrSum(float* A, float* B, float* res, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int width = blockDim.y * gridDim.y;

    res[i * width + j] = A[i * width + j] + B[i * width + j];
}

int main() {
    //step 1
    int n = 128;
    int m = 64;
    float* h_A = new float[n*m];
    float* h_B = new float[n*m];
    float* h_res = new float[n*m];

    //step 2
    float* d_A;
    float* d_B;
    float* d_res;
    int nbytes = n * m * sizeof(float);
    hipMalloc(&d_A, nbytes);
    hipMalloc(&d_B, nbytes);
    hipMalloc(&d_res, nbytes);

    //
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            h_A[i*m + j] = 1;
            h_B[i*m + j] = 2;
        }
    }

    //step 3
    hipMemcpy(d_A, h_A, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nbytes, hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    //step 4
    hipEventRecord(start);
    dim3 num_blocks(16, 16);
    dim3 block_size(8, 4);
    // необходимо num_blocks.x * block_size.x = n,  num_blocks.y * block_size.y = m
    ArrSum<<<num_blocks, block_size>>>(d_A, d_B, d_res, n * m);

    hipEventRecord(end);

    //step 5
    hipMemcpy(h_res, d_res, nbytes, hipMemcpyDeviceToHost);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    std::cout << "Time elapsed: " << ms << " ms " << std::endl;

    /*
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
            std::cout << h_res[i*m + j] << " ";
        std::cout << std::endl;
    }
    //*/

    //step 6
    delete[] h_A;
    delete[] h_B;
    delete[] h_res;
    //step 7
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_res);
}