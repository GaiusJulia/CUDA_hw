
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

// Задание 3: Перемножение матрицы на вектор
__global__
void ArrVectMul(float* A, float* x, float* res, int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    res[i] = 0.;
    for (int k = 0; k < size; k++)
    {
        res[i] += A[i * size + k] * x[k];
        //printf("A[%d][%d] = %f \n", i, k, A[i * size + k]);
        //printf("x[%d] = %f \n", k, x[k]);
    }

}

int main() {
    int n = 10;
    int m = 20;
    float* h_A = new float[n*m];
    float* h_x = new float[m];
    float* h_res = new float[n];

    //step 2
    float* d_A;
    float* d_x;
    float* d_res;
    int bytes = sizeof(float);
    hipMalloc(&d_A, n * m * bytes);
    hipMalloc(&d_x, m * bytes);
    hipMalloc(&d_res, n * bytes);

    //
    for (int i = 0; i < m; i++)
    {
        h_x[i] = 1;

        for (int j = 0; j < n; j++)
        {
            h_A[j*m + i] = j;
        }
    }

    //step 3
    hipMemcpy(d_A, h_A, n * m * bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, m *bytes, hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    //step 4
    hipEventRecord(start);
    int num_blocks = 16;
    int block_size = 16;
    ArrVectMul<<<num_blocks, block_size>>>(d_A, d_x, d_res, m);

    hipEventRecord(end);

    //step 5
    hipMemcpy(h_res, d_res, n * bytes, hipMemcpyDeviceToHost);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    std::cout << "Time elapsed: " << ms << " ms " << std::endl;

    
    for (int i = 0; i < n; i++)
    {
        std::cout << h_res[i] << " ";
    }

    //step 6
    delete[] h_A;
    delete[] h_x;
    delete[] h_res;
    //step 7
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_res);
}
