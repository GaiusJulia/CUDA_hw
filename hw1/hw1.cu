
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>

// Задание 1: поэлементное перемножение векторов
__global__
void KernelMul(int n, float* x, float* y, float* res){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int index = tid; index < n; index += stride){
        res[index] = x[index] * y[index];
    }
}

int main() {
    //step 1
    int n = 1 << 20;
    float* h_x = new float[n];
    float* h_y = new float[n];
    float* h_res = new float[n];

    //step 2
    float* d_x;
    float* d_y;
    float* d_res;
    int nbytes = n * sizeof(float);
    hipMalloc(&d_x, nbytes);
    hipMalloc(&d_y, nbytes);
    hipMalloc(&d_res, nbytes);

    //
    for (int i = 0; i < n; i++)
    {
        h_x[i] = 2 * i;
        h_y[i] = 3 * i;
    }

    //step 3
    hipMemcpy(d_x, h_x, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, nbytes, hipMemcpyHostToDevice);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    //step 4
    hipEventRecord(start);

    // первое число - число блоков, второе число - BLOCKSIZE (число потоков в блоке)
    KernelMul<<<1, 2>>>(n, d_x, d_y, d_res);

    hipEventRecord(end);

    //step 5
    hipMemcpy(h_res, d_res, nbytes, hipMemcpyDeviceToHost);
    hipEventSynchronize(end);

    float ms;
    hipEventElapsedTime(&ms, start, end);
    std::cout << "Time elapsed: " << ms << " ms " << std::endl;

    /*
    for (int i = 0; i < n; i++)
    {
        std::cout << h_res[i] << " ";
    }
    //*/

    //step 6
    delete[] h_x;
    delete[] h_y;
    delete[] h_res;
    //step 7
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
}
